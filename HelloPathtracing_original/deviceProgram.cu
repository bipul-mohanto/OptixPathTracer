#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <random.h>

#include <cuda/helpers.h>

#include <sutil/vec_math.h>

#include "LaunchParams.h"
  
#include "Disney.cuh"

//#define USE_JITTERED_UNIFORM
#define USE_STRATIFIED
#define kProbeSamples 1.f
#define kBsdfSamples 1.f

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" {
     __constant__ LaunchParams params;
}

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

const int RAY_STATE_FLAGS_DONE               = 1 << 0;
const int RAY_STATE_FLAGS_SECONDARY_RAY      = 1 << 1;
const int RAY_STATE_FLAGS_SHADOW_RAY         = 1 << 2;

struct RadiancePRD
{
    float3       radiance;
    float3       alpha;

    float3       origin;
    float3       direction;

    float3       normal;
    float3       albedo;

    float4       lightSamples;

    float bsdfPdf = 1.0f;
    float3 pathThroughput;
    float rayEta = 1.0f;
    float3 rayAbsorption;
    BSDFType rayType = eReflected;    

    int depth;
    int stateFlags = 0;

    unsigned int seed;
    Random       rand;
};


struct Onb
{
    __forceinline__ __device__ Onb(const float3& normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1)
{
    const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}


static __forceinline__ __device__ void  packPointer(void* ptr, unsigned int& i0, unsigned int& i1)
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}


static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}

static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    RadiancePRD* prd
)
{
    // TODO: deduce stride from num ray-types passed in params

    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1);
}


static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax
)
{
    unsigned int occluded = 0u;
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                    // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,      // SBT offset
        RAY_TYPE_COUNT,          // SBT stride
        RAY_TYPE_OCCLUSION,      // missSBTIndex
        occluded);
    return occluded;
}
  
extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */ }

extern "C" __global__ void __miss__radiance()
{
    RadiancePRD* prd = getPRD<RadiancePRD>();
    const float3 ray_dir = optixGetWorldRayDirection();

    /*float weight = 1.0f;
    // probability that this dir was already sampled by probe sampling
    float skyPdf = ProbePdf(params.probe, ray_dir);

    int N = kProbeSamples + kBsdfSamples;
    float cbsdf = kBsdfSamples / N;
    float csky = float(kProbeSamples) / N;

    weight = cbsdf * prd->bsdfPdf / (cbsdf * prd->bsdfPdf + csky * skyPdf);

    prd->radiance += weight * make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_dir))) * prd->pathThroughput;*/
    prd->albedo = make_float3(0.f);
    prd->normal = make_float3(0.f);

    if ((prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) != 0) {
        const float3 ray_dir = optixGetWorldRayDirection();
        //prd->alpha = make_float3(1.0f);
        //prd->radiance = make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_dir)));
    }

    prd->stateFlags |= RAY_STATE_FLAGS_DONE;
}

extern "C" __global__ void __anyhit__occlusion()
{ 
    setPayloadOcclusion(true);
}

extern "C" __global__ void __closesthit__occlusion()
{ 
    
}

extern "C" __global__ void __miss__occlusion()
{ 
    setPayloadOcclusion(false);
}

static __device__ __forceinline__ float3 SampleLights(const Material& material, float3 albedo, const float etaI, const float etaO, const float3& surfacePos, const float3& surfaceNormal, const float3& wo, Random& rand)
{
    float3 sum = make_float3(0.0f);

    for (int i = 0; i < kProbeSamples; ++i)
    {
        float3 skyColor;
        float skyPdf;
        float3 wi;
        
        ProbeSample(params.probe, wi, skyColor, skyPdf, rand);

        const bool occluded = traceOcclusion(
            params.traversable,
            surfacePos,
            wi,
            0.01f,         // tmin
            1e16f  // tmax
        );

        if (!occluded)
        {
            float bsdfPdf = BSDFPdf(material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
            float3 f = BSDFEval(material, albedo, etaI, etaO, surfacePos, surfaceNormal, wo, wi);

            if (bsdfPdf > 0.0f)
            {
                int N = kProbeSamples + kBsdfSamples;
                float cbsdf = kBsdfSamples / N;
                float csky = float(kProbeSamples) / N;
                float weight = csky * skyPdf / (cbsdf * bsdfPdf + csky * skyPdf);

                if (weight > 0.0f) {
                    float3 val = weight * skyColor * f * abs(dot(wi, surfaceNormal)) / skyPdf * (1.0f / kProbeSamples);
                    sum += val;// make_float3((val.x + val.y + val.z) / 3.0f);
                }
            }
        }
    }
    return sum;
}

static __device__ __forceinline__ float3 SampleShadow(const Material& material, float3 albedo, const float etaI, const float etaO, const float3& surfacePos, const float3& surfaceNormal, const float3& wo, Random& rand)
{
    float3 sum = make_float3(0.0f);

    for (int i = 0; i < kProbeSamples; ++i)
    {
        float3 skyColor;
        float skyPdf;
        float3 wi;

        ProbeSample(params.probe, wi, skyColor, skyPdf, rand);

        const bool occluded = traceOcclusion(
            params.traversable,
            surfacePos,
            wi,
            0.01f,         // tmin
            1e16f  // tmax
        );

        if (occluded)
        {
            float bsdfPdf = BSDFPdf(material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
            float3 f = BSDFEval(material, albedo, etaI, etaO, surfacePos, surfaceNormal, wo, wi);

            if (bsdfPdf > 0.0f)
            {
                int N = kProbeSamples + kBsdfSamples;
                float cbsdf = kBsdfSamples / N;
                float csky = float(kProbeSamples) / N;
                float weight = csky * skyPdf / (cbsdf * bsdfPdf + csky * skyPdf);

                if (weight > 0.0f) {
                    float3 val = weight * skyColor * f * abs(dot(wi, surfaceNormal)) / skyPdf * (1.0f / kProbeSamples);
                    sum += val;
                }
            }
        }
    }
    return sum;
}


//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    const int    w = params.frame.size.x;
    const int    h = params.frame.size.y;
    const float3 eye = params.camera.eye;
    const float3 U = params.camera.U;
    const float3 V = params.camera.V;
    const float3 W = params.camera.W;
    const uint3  idx = optixGetLaunchIndex();
    const unsigned int    subframe_index = params.frame.subframe_index;
      
    float3 result = make_float3(0.0f);
    //bm
    //int samples_per_launch = (subframe_index == 0) ? 4 : params.samples_per_launch;
    int samples_per_launch = params.samples_per_launch;
    int i = samples_per_launch;

    unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

    float3 normal = make_float3(0.f);
    float3 albedo = make_float3(0.f);
    float3 alpha = make_float3(0.f);    

    float3 backplate = make_float3(0.f);   

    do
    {        
        float3 directLight = make_float3(0.0f);
        float3 indirectLight = make_float3(0.0f);

        RadiancePRD prd;
        
        prd.radiance = make_float3(0.f);        
        prd.alpha = make_float3(0.f);

        prd.seed = seed;
        prd.rand = Random(seed);

        prd.rayEta = 1.0f;
        prd.pathThroughput = make_float3(1.f);
        prd.rayAbsorption = make_float3(0.0f);
        prd.bsdfPdf = 1.0f;
        prd.normal = make_float3(0.0f);
        prd.albedo = make_float3(0.0f);
        prd.stateFlags = 0;
        prd.depth = 0;

        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
        ) - 1.0f;        

        /*float3 ray_origin;
        if (idx.x < w / 2.0f)
        {
            ray_origin = eye - U * 0.1f;
            d = d + make_float2(0.25f, 0.f);
        } else {
            ray_origin = eye + U * 0.1f;
            d = d - make_float2(0.25f, 0.f);
        }*/

        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;         

        

        backplate = make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_direction)));

        for (;; )
        {
            prd.radiance = make_float3(0.f);

            traceRadiance(
                params.traversable,
                ray_origin,
                ray_direction,
                0.001f,  // tmin       // TODO: smarter offset
                1e16f,  // tmax
                &prd);

            if (prd.depth == 0.f) {
                normal += prd.normal;
                albedo += prd.albedo;               
            }           

            if ((prd.stateFlags & RAY_STATE_FLAGS_DONE) || prd.depth >= 8) // TODO RR, variable for depth
                break;

            if (prd.depth == 0) {
                directLight += prd.radiance;
            }
            else {
                indirectLight += prd.radiance;
            }

            ++prd.depth;

            ray_origin = prd.origin;
            ray_direction = prd.direction;                        
        }

        result += directLight + indirectLight;
        alpha += prd.alpha;       

    } while (--i);

    normal /= static_cast<float>(samples_per_launch);
    albedo /= static_cast<float>(samples_per_launch);
    alpha /= static_cast<float>(samples_per_launch);

    float3 color = (backplate * static_cast<float>(params.samples_per_launch)) * (1.0f - alpha) + result;

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.frame.size.x + launch_index.x;
    float3         accum_color = color / static_cast<float>(params.samples_per_launch); // result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        accum_color = clamp(accum_color, make_float3(0.0), make_float3(10.0f));
        const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.frame.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.frame.accum_buffer[image_index] = make_float4(accum_color, 1.0f);

    //params.frame.frame_buffer[image_index] = make_color(accum_color);
    params.frame.frame_buffer[image_index] = make_color(accum_color);

    params.frame.normal_buffer[image_index] = make_float4(normal, 1.0f);
    params.frame.color_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame.albedo_buffer[image_index] = make_float4(albedo, 1.0f);
}

extern "C" __global__ void __closesthit__radiance()
{
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    const int    prim_idx = optixGetPrimitiveIndex();
    const float3 ray_dir = optixGetWorldRayDirection();

    const uint3 index = sbtData.index[prim_idx];

    const float3 v0 = sbtData.vertex[index.x];
    const float3 v1 = sbtData.vertex[index.y];
    const float3 v2 = sbtData.vertex[index.z];
    const float3 N_0 = normalize(cross(v1 - v0, v2 - v0));

    float3 N = faceforward(N_0, -ray_dir, N_0);

    const float t = optixGetRayTmax();
    const float rayTime = 0.f;
    const float3 P = optixGetWorldRayOrigin() + t * ray_dir;    

    float outEta;
    float3 outAbsorption;

    RadiancePRD* prd = getPRD<RadiancePRD>();

    if ((sbtData.material.flags & MATERIAL_FLAG_SHADOW_CATCHER) != 0 && (prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) != 0) {
        prd->origin = P;
        prd->direction = ray_dir;
        --prd->depth;
        return;
    }

    prd->normal = N;
    prd->albedo = sbtData.material.color;
    if (sbtData.hasTexture && sbtData.texcoord) {
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        const float2 tc = (1.f - u - v) * sbtData.texcoord[index.x]
            + u * sbtData.texcoord[index.y]
            + v * sbtData.texcoord[index.z];

        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        prd->albedo = make_float3(fromTexture);
        //prd->albedo = make_float3(1.f,0.f,0.f);
    }
    float& bsdfPdf = prd->bsdfPdf;    

    if (prd->rayEta == 1.0f)
    {
        outEta = sbtData.material.GetIndexOfRefraction();
        outAbsorption = sbtData.material.absorption;
    }
    else
    {
        // returning to free space
        outEta = 1.0f;
        outAbsorption = make_float3(0.0f);
    }

    // update throughput based on absorption through the medium
    //prd->pathThroughput *= exp(-prd->rayAbsorption * rayTime);    

    //float3 lightValue = SampleLights(sbtData.material, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
    //prd->radiance += prd->pathThroughput * clamp(lightValue, make_float3(0), make_float3(2));

    if((sbtData.material.flags & MATERIAL_FLAG_SHADOW_CATCHER) == 0){
        float3 lightSample = SampleLights(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
        prd->radiance += prd->pathThroughput * lightSample;
        prd->alpha = make_float3(1.0f);
    }
    else {
        float3 shadowSample = SampleShadow(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
        prd->alpha += prd->pathThroughput * shadowSample;
    }

    //float3 lightSample = SampleShadow(sbtData.material, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
    //prd->radiance += prd->pathThroughput * lightSample;

    
    if ((prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) == 0) {
        prd->radiance += sbtData.material.emission;   
    }    

    float3 u, v;
    BasisFromVector(N, &u, &v);

    float3 bsdfDir;
    BSDFType bsdfType;

    BSDFSample(sbtData.material, prd->rayEta, outEta, P, u, v, N, -ray_dir, bsdfDir, bsdfPdf, bsdfType, prd->rand);

    if (bsdfPdf <= 0.0f){
        prd->stateFlags |= RAY_STATE_FLAGS_DONE;
        return;
    }

    // reflectance
    float3 f = BSDFEval(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, bsdfDir);

    // update ray medium if we are transmitting through the material
    if (dot(bsdfDir, N) <= 0.0f)
    {
        prd->rayEta = outEta;
        prd->rayAbsorption = outAbsorption;
    }

    // update throughput with primitive reflectance
    prd->pathThroughput *= f * abs(dot(N, bsdfDir)) / bsdfPdf;

    // update ray direction and type
    prd->rayType = bsdfType;
    prd->direction = bsdfDir;
    prd->origin = P;

    prd->stateFlags |= RAY_STATE_FLAGS_SECONDARY_RAY;
}
