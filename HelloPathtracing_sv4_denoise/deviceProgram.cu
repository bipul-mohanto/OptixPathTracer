#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
//======================================================================== //
//bm, for better cuda code syntax visualization
#include "hip/hip_runtime.h"
#include<> 

#include <optix_device.h>

#include <random.h>
#include <cuda/helpers.h>

// in-folder
#include <sutil/vec_math.h>
#include "LaunchParams.h"
#include "Disney.cuh"
#include "maths.h"

// bm: these two has no effect at all
#define USE_JITTERED_UNIFORM
#define USE_STRATIFIED
// check this two in Disney.cuh and sample.h

#define kProbeSamples 1.f //k stands for constant? 
#define kBsdfSamples 1.f // why value limited to 1.0

// global variables for the ray generation program
__device__ float tmin {0.01f}; // instead of 0.0f
__device__ float tmax {1e16f}; // instead of 1e27f

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" {
     __constant__ LaunchParams params;
}

//--------------------------------------------------------------------------
//
//
//---------------------------------------------------------------------------
const int RAY_STATE_FLAGS_DONE               = 1 << 0;//0, why
const int RAY_STATE_FLAGS_SECONDARY_RAY      = 1 << 1;//1, why
const int RAY_STATE_FLAGS_SHADOW_RAY         = 1 << 2;//2, why
//bm: the secondary ray is the shadow ray, no use of this value at this moment 

struct RadiancePRD //?
{
    float3       radiance;
    float3       alpha;

    float3       origin;
    float3       direction;

    //bm: these two for ai denoising
    float3       normal;
    float3       albedo;

    //bm: no use, comment
    //float4       lightSamples;

    float bsdfPdf = 0.0f;    // why??? value was 0.0f
    float3 pathThroughput ; // why?
    float rayEta = 0.0f; //why ??? value was 1.0f
    float3 rayAbsorption; // why?
    BSDFType rayType = eReflected;    
    //Disney, 
    //bm no differences between eTransmitted and eRefracted
    
    int depth;
    int stateFlags = 0;

    unsigned int seed;
    Random       rand;
};


struct Onb // what is Onb? something related to normal, 
{
    __forceinline__ __device__ Onb(const float3& normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

// bm
static __forceinline__ __device__ float3 reinhardToneMap(const float3& color, const float white)
{
    const float luminance = 0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z;//luminance
    return (color * 1.0f) / (1.0f + luminance / white);

}
// bm: not working at this moment (7.6.2023)
static __forceinline__ __device__ float3 gaussianFilter(float3 color, float kernelSize, float sigma)
{
	float3 result = make_float3(0.0f);
	float sum = 0.0f;
	for (int x = -kernelSize; x <= kernelSize; ++x)
	{
		for (int y = -kernelSize; y <= kernelSize; ++y)
		{
			float2 offset = make_float2(x, y);
			float weight = (1/(2*M_PI*sigma*sigma)) * expf(-(offset.x * offset.x + offset.y * offset.y) / (2.0f * sigma * sigma));
			result += weight * color;
			//sum += weight;
		}
	}
	return result / sum;
}


static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1)
{
    const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}


static __forceinline__ __device__ void  packPointer(void* ptr, unsigned int& i0, unsigned int& i1)
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

//!bm: no use, commenting now 
//! // where is the use?
/*
static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}
*/

static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    RadiancePRD* prd
)
{
    // TODO: deduce stride from num ray-types passed in params (???)

    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                      // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1);
}


static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax
)
{
    unsigned int occluded = 0u;
    //The acceleration structure (AS) traversal is started with an optixTrace call
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                    // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,//OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,      // SBT offset
        RAY_TYPE_COUNT,          // SBT stride
        RAY_TYPE_OCCLUSION,      // missSBTIndex
        occluded);
    return occluded;
}
  
extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */ }

extern "C" __global__ void __miss__radiance()
{
    RadiancePRD* prd = getPRD<RadiancePRD>();
    const float3 ray_dir = optixGetWorldRayDirection();

    //bm: already was commented 
    /*float weight = 1.0f;
    // probability that this dir was already sampled by probe sampling
    float skyPdf = ProbePdf(params.probe, ray_dir);

    int N = kProbeSamples + kBsdfSamples;
    float cbsdf = kBsdfSamples / N;
    float csky = float(kProbeSamples) / N;

    weight = cbsdf * prd->bsdfPdf / (cbsdf * prd->bsdfPdf + csky * skyPdf);

    prd->radiance += weight * make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_dir))) * prd->pathThroughput;*/

    prd->albedo = make_float3(0.f);
    prd->normal = make_float3(0.f);

    if ((prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) != 0) {
        const float3 ray_dir = optixGetWorldRayDirection();
        // bm: previously commented
        //prd->alpha = make_float3(1.0f);
        //prd->radiance = make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_dir)));
    }

    prd->stateFlags |= RAY_STATE_FLAGS_DONE;
}

extern "C" __global__ void __anyhit__occlusion()
{ 
    setPayloadOcclusion(true);
}

extern "C" __global__ void __closesthit__occlusion()
{ 
    
}

extern "C" __global__ void __miss__occlusion()
{ 
    setPayloadOcclusion(false);
    // bm: occlusion implementation missing???
    // true not working

}

//! bm: light sampling? explicit or implicit?
static __device__ __forceinline__ float3 SampleLights(const Material& material, float3 albedo, const float etaI, const float etaO, const float3& surfacePos, const float3& surfaceNormal, const float3& wo, Random& rand)
{
    float3 sum = make_float3(0.0f);

    for (int i = 0; i < kProbeSamples; ++i)
    {
        float3 skyColor;
        float skyPdf;
        float3 wi;
        
        ProbeSample(params.probe, wi, skyColor, skyPdf, rand);

        const bool occluded = traceOcclusion(
            params.traversable,
            surfacePos,
            wi,
            tmin,//0.01f,         // tmin
            tmax//1e16f  // tmax
        );

        if (!occluded)
        {
            float bsdfPdf = BSDFPdf(material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
            float3 f = BSDFEval(material, albedo, etaI, etaO, surfacePos, surfaceNormal, wo, wi);

            if (bsdfPdf > 0.0f)
            {
                int N = kProbeSamples + kBsdfSamples;
                float cbsdf = kBsdfSamples / N;
                float csky = float(kProbeSamples) / N;
                float weight = csky * skyPdf / (cbsdf * bsdfPdf + csky * skyPdf);

                if (weight > 0.0f) {
                    float3 val = weight * skyColor * f * abs(dot(wi, surfaceNormal)) / skyPdf * (1.0f / kProbeSamples);
                    sum += val;// make_float3((val.x + val.y + val.z) / 3.0f);
                }
            }
        }
    }
    return sum;
}

//! Random?
static __device__ __forceinline__ float3 SampleShadow(const Material& material, float3 albedo, const float etaI, const float etaO, const float3& surfacePos, const float3& surfaceNormal, const float3& wo, Random& rand)
{
    float3 sum = make_float3(0.0f);

    for (int i = 0; i < kProbeSamples; ++i)
    {
        float3 skyColor;
        float skyPdf;
        float3 wi;

        ProbeSample(params.probe, wi, skyColor, skyPdf, rand);

        const bool occluded = traceOcclusion(
            params.traversable,
            surfacePos,
            wi,
            tmin,//0.01f,         // tmin
            tmax//1e16f  // tmax
        );

        if (occluded)
        {
            float bsdfPdf = BSDFPdf(material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
            float3 f = BSDFEval(material, albedo, etaI, etaO, surfacePos, surfaceNormal, wo, wi);

            if (bsdfPdf > 0.0f)
            {
                int N = kProbeSamples + kBsdfSamples;
                float cbsdf = kBsdfSamples / N;
                float csky = float(kProbeSamples) / N;
                float weight = csky * skyPdf / (cbsdf * bsdfPdf + csky * skyPdf);

                if (weight > 0.0f) {
                    float3 val = weight * skyColor * f * abs(dot(wi, surfaceNormal)) / skyPdf * (1.0f / kProbeSamples);
                    sum += val;
                }
            }
        }
    }
    return sum;
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here, all the shaders gradually
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    const int    w = params.frame.size.x;
    const int    h = params.frame.size.y;
    const float3 eye = params.camera.eye;
    const float3 U = params.camera.U;
    const float3 V = params.camera.V;
    const float3 W = params.camera.W;
    uint3  idx = optixGetLaunchIndex();

    const unsigned int    subframe_index = params.frame.subframe_index; // what is subframe index doing? accumulation?
    // bm: this was wrong, made foveated region 4spp, replaced with next statement, before i value  
    //int samples_per_launch = (subframe_index == 0) ? 4 : params.samples_per_launch;

    int samples_per_launch = params.samples_per_launch;
    int i = samples_per_launch;//why I need previous line?

//! ------------------------- random seed generator
     unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

// bm: others, tea<4> however is the best so far, later will come back to this
//unsigned int cc = idx.y * w + idx.x;
//unsigned int seed = lcg2(cc);
//unsigned int seed = lcg2(cc);
//unsigned int seed = rnd(cc);
//unsigned int seed = rot_seed(cc, subframe_index); 
//-------------------------------------------------------
    float3 result = make_float3(0.0f);

    const uint2    launch_index = make_uint2(optixGetLaunchIndex());
    idx = idx * params.frame.factor + make_uint3(params.frame.offset, 0);

    float range = length(make_float3(idx) - make_float3(make_uint3(params.frame.c, 0)));

    if (range < params.frame.r_inner || range > params.frame.r_outer) {
        return;

    }
        
    // bm: requires for denoising, else no use I see so far
    float3 normal = make_float3(0.f);
    float3 albedo = make_float3(0.f); //denoiser
    float3 alpha = make_float3(0.f);  //denoiser  

    float3 backplate = make_float3(0.f);   //result? then what is result?
    do
    {        
        float3 directLight = make_float3(0.0f); //original value;0 bm: can add effect on result
        float3 indirectLight = make_float3(0.0f);//original value;0

        RadiancePRD prd;
        
        prd.radiance = make_float3(0.f); //bm: has effect         
        prd.alpha = make_float3(0.f); // 1 makes the background light dark
//!------------------------------------------sampling pattern? random?
        prd.seed = seed;
        prd.rand = Random(seed);
//--------------------------------------------------------
        prd.rayEta = 1.0f;// bm: original value: 1 was previous value
        prd.pathThroughput = make_float3(1.f); //1 why??? 
        prd.rayAbsorption = make_float3(0.f);// 0 bm: effect has
        prd.bsdfPdf = 1.0f;
        prd.normal = make_float3(0.0f);
        prd.albedo = make_float3(0.0f);
        prd.stateFlags = 0;
        prd.depth=0; // what is this depth? why only 0-3 values are working? 
        //original value was 1

//!---------------------------------- anti-aliasing
        // The center of each pixel is at fraction (0.5,0.5) 
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));
        
#define USE_ANTIALIASING 1
#ifdef USE_ANTIALIASING
        float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
        ) - 1.0f;
#else
        float2 d = 2.0f * make_float2(static_cast<float>(idx.x) / static_cast<float>(w), static_cast<float>(idx.y) / static_cast<float>(h)) - 1.0f;
#endif // USE_ANTIALIASING

        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;         

        backplate = make_float3(ProbeEval(params.probe, ProbeDirToUV(ray_direction)));

        for (;; )
        {
            prd.radiance = make_float3(0.f);

            traceRadiance(
                params.traversable,
                ray_origin,
                ray_direction,
                tmin, //0.001f,  // tmin       // TODO: smarter offset
                tmax,//1e16f,  // tmax
                &prd);
            
            if (prd.depth == 0.f) {
                normal += prd.normal;
                albedo += prd.albedo;               
            }           
            //! bm: ray bounce termination (VVI)
            
            if ((prd.stateFlags & RAY_STATE_FLAGS_DONE) || prd.depth >=4) //no effect with bounce
                break;
         
            //!TODO RR, variable for depth

//!TODO: Russian Roullet
 
            if (prd.depth == 0) {
                directLight += prd.radiance;
            }
            else {
                indirectLight += prd.radiance;
            }

            ++prd.depth;
            
            ray_origin = prd.origin;
            ray_direction = prd.direction;                        
            
        }

        result += directLight + indirectLight;// bm: light amplification
        alpha += prd.alpha;       

    } while (--i);

    normal /= static_cast<float>(samples_per_launch);
    albedo /= static_cast<float>(samples_per_launch);
    alpha /= static_cast<float>(samples_per_launch);

//! sv (foveation related)
    for (int i = 0; i < params.frame.fillSize; ++i) {
        for (int j = 0; j < params.frame.fillSize; ++j) {
 
            const uint3 launch_index = optixGetLaunchIndex()*params.frame.factor;
            uint2 index = make_uint2(
                launch_index.x + i + params.frame.offset.x,
                launch_index.y + j + params.frame.offset.y);

            index = clamp(index, make_uint2(0, 0), make_uint2(w-1, h-1));
            
            const unsigned int image_index = (index.y) * w + (index.x);            

            float3 color = (backplate * static_cast<float>(params.samples_per_launch)) * (1.0f - alpha) + result;

            float3 accum_color = color / static_cast<float>(params.samples_per_launch); // result / static_cast<float>(params.samples_per_launch);

//! accumulation 
//! sv 
             
            if (subframe_index > 0 && !params.frame.redraw)// frame==0, allow per frame rendering
            {
                accum_color = clamp(accum_color, make_float3(0.0), make_float3(10.0f)); //10.0f, what is this doing?
                const float                 alpha_value = 1.0f / static_cast<float>(subframe_index + 1);
                const float3 accum_color_prev = make_float3(params.frame.accum_buffer[image_index]);
                accum_color = lerp(accum_color_prev, accum_color, alpha_value);
               
            }
            params.frame.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
#define EXPOSURE_CORR_ON
#ifdef EXPOSURE_CORR_ON
            float3 pprocessingExposureCorrected = float3(accum_color * pow(2.0f, 4.0f)) ; // bm: accum_color is the output final
            // pow(2.0f, 2.0f) // original
            //!TODO: interaction time is not doing the job, why?
            //params.frame.frame_buffer[image_index] = make_color(pprocessingExposureCorrected);
#else
            float3 pprocessingExposureCorrected = float3(accum_color);
            // result in very dim, bounce and sample number has no effect, very probably, the miss rays (those not               recach the light) has 0 contribution, that is the reason, bettwe with exposure correction
#endif

#define TONE_MAPPING__ENABLE
#ifdef TONE_MAPPING__ENABLE
            params.frame.frame_buffer[image_index] = make_color(reinhardToneMap(pprocessingExposureCorrected, 1.0f));
#else
            params.frame.frame_buffer[image_index] = make_color(pprocessingExposureCorrected);
#endif // TONE_MAPPING

//TODO: denoising
#define GAUSSIAN_OFF
#ifdef GAUSSIAN_ON
            params.frame.frame_buffer[image_index] = make_color(gaussianFilter(pprocessingExposureCorrected, 3.0f, 10.f));
#else
            params.frame.frame_buffer[image_index] = make_color(pprocessingExposureCorrected);
#endif

           
// bm: these buffers only for denoising part
            // params.frame.normal_buffer[image_index] = make_float4(normal, 1.0f);
            // params.frame.color_buffer[image_index] = make_float4(accum_color, 1.0f) ; //replaced with alpha 0, no effect, why?
            // params.frame.albedo_buffer[image_index] = make_float4(albedo, 1.0f);
        }
    }
}

extern "C" __global__ void __closesthit__radiance()
{
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    const int    prim_idx = optixGetPrimitiveIndex();
    const float3 ray_dir = optixGetWorldRayDirection();

    const uint3 index = sbtData.index[prim_idx];

    const float3 v0 = sbtData.vertex[index.x];
    const float3 v1 = sbtData.vertex[index.y];
    const float3 v2 = sbtData.vertex[index.z];
    const float3 N_0 = normalize(cross(v1 - v0, v2 - v0)); //the great normal vector 

    float3 N = faceforward(N_0, -ray_dir, N_0);

    const float t = optixGetRayTmax();
    const float rayTime = 0.0f;
    const float3 P = optixGetWorldRayOrigin() + t * ray_dir;    

    float outEta;
    float3 outAbsorption;

    RadiancePRD* prd = getPRD<RadiancePRD>();

    // bm: what is happening here?
    if ((sbtData.material.flags & MATERIAL_FLAG_SHADOW_CATCHER) != 0 && (prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) != 0) {
        prd->origin = P;
        prd->direction = ray_dir;
        --prd->depth;
        return;
    }

    prd->normal = N;
    prd->albedo = sbtData.material.color;
    if (sbtData.hasTexture && sbtData.texcoord) {
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        const float2 tc = (1.f - u - v) * sbtData.texcoord[index.x]
            + u * sbtData.texcoord[index.y]
            + v * sbtData.texcoord[index.z];

        // bm: why? ans:  cuda function
        float4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y); //
        prd->albedo = make_float3(fromTexture);
//        if (prd->albedo = make_float3(0.0f)) {
//            prd->albedo = make_float3(1.f, 0.f, 1.f); // magenta color
           //bm: sanity check if texture error
//        }
    }
    float& bsdfPdf = prd->bsdfPdf;    

    if (prd->rayEta == 1.0f)
    {
        outEta = sbtData.material.GetIndexOfRefraction();
        outAbsorption = sbtData.material.absorption;
    }
    else
    {
        // returning to free space
        outEta = 1.0f;
        outAbsorption = make_float3(0.0f);
    }

    if((sbtData.material.flags & MATERIAL_FLAG_SHADOW_CATCHER) == 0){
        float3 lightSample = SampleLights(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
        prd->radiance += prd->pathThroughput * lightSample;
        prd->alpha = make_float3(1.0f);
    }
    else {
        float3 shadowSample = SampleShadow(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, prd->rand);
        prd->alpha += prd->pathThroughput * shadowSample;
    }
   
    if ((prd->stateFlags & RAY_STATE_FLAGS_SECONDARY_RAY) == 0) {
        prd->radiance += sbtData.material.emission;   
    }    

    float3 u, v;
    BasisFromVector(N, &u, &v);

    float3 bsdfDir;
    BSDFType bsdfType;

    BSDFSample(sbtData.material, prd->rayEta, outEta, P, u, v, N, -ray_dir, bsdfDir, bsdfPdf, bsdfType, prd->rand);

    if (bsdfPdf <= 0.0f){
        prd->stateFlags |= RAY_STATE_FLAGS_DONE;
        return;
    }

    // reflectance
    float3 f = BSDFEval(sbtData.material, prd->albedo, prd->rayEta, outEta, P, N, -ray_dir, bsdfDir);

    // update ray medium if we are transmitting through the material
    if (dot(bsdfDir, N) <= 0.0f)
    {
        prd->rayEta = outEta;
        prd->rayAbsorption = outAbsorption;
    }

    // update throughput with primitive reflectance
    prd->pathThroughput *= f * abs(dot(N, bsdfDir)) / bsdfPdf;

    // update ray direction and type
    prd->rayType = bsdfType;
    prd->direction = bsdfDir;
    prd->origin = P;

    prd->stateFlags |= RAY_STATE_FLAGS_SECONDARY_RAY;
}
